#include "hip/hip_runtime.h"
#include <cudasks.cuh>

int h_in[LEN*LEN];

int* d_in;
int* d_out;

__device__ int getIndex(int x, int y, int w, int h) {
	if (x>=0) x %= w;	else x += w - ((-x)%w);
	if (y>=0) y %= h;	else y += h - ((-y)%h);
	return w*y + x;
}

__global__ void gameOfLife(int* d_in, int* d_out) {
	int x = threadIdx.x, y = blockIdx.x, w = blockDim.x, h = gridDim.x;

	int n00 = getIndex(x-1, y-1, w, h), n10 = getIndex(x  , y-1, w, h), n20 = getIndex(x+1, y-1, w, h),
		n01 = getIndex(x-1, y  , w, h), idx = getIndex(x  , y  , w, h), n21 = getIndex(x+1, y  , w, h),
		n02 = getIndex(x-1, y+1, w, h), n12 = getIndex(x  , y+1, w, h), n22 = getIndex(x+1, y+1, w, h);
	
	int lif = d_in[n00] + d_in[n10] + d_in[n20]
			+ d_in[n01] 			+ d_in[n21]
			+ d_in[n02] + d_in[n12] + d_in[n22];
	
	int cell = d_in[idx];

	if (cell>0) {
		if (lif==2 || lif==3)
			d_out[idx] = 1;
		else d_out[idx] = 0;
	} else {
		if (lif==3)
			d_out[idx] = 1;
		else d_out[idx] = 0;
	}

	//printf("(%d -> %d | l=%d)\n", cell, d_out[idx], lif);
}

__global__ void singleRotation(int* d_in, int* d_out, int paircut) {
	int x = 2*threadIdx.x+paircut,	y = 2*blockIdx.x+paircut,
		w = 2*blockDim.x,	h = 2*gridDim.x;

	int n00 = getIndex(x  , y  , w, h), n10 = getIndex(x+1, y  , w, h),
		n01 = getIndex(x  , y+1, w, h), n11 = getIndex(x+1, y+1, w, h);
	
	int lif = d_in[n00] + d_in[n10] + d_in[n01] + d_in[n11];

	//	00 10 | 01 00
	//	01 11 |	11 10
	if (lif==1) {
		d_out[n00] = d_in[n01];
		d_out[n10] = d_in[n00];
		d_out[n01] = d_in[n11];
		d_out[n11] = d_in[n10];
	}
}

int paircut = 0;
void triggerPaircut() {
	paircut = (paircut) ? 0 : 1;
}

void initArrays() {
	for (int j = BORDER; j<LEN-BORDER; j++) {
		for (int i = BORDER; i<LEN-BORDER; i++) {
			h_in[j*LEN+i] = (rand()%1000 > 800) ? 1 : 0;
		}
	}

	hipMalloc(&d_in, B2LEN);
	hipMalloc(&d_out, B2LEN);
}

void calculateLife(int N) {
	for (int i = 0; i<N; i++) {
		hipMemcpy(d_in, h_in, B2LEN, hipMemcpyHostToDevice);
		// gameOfLife<<<LEN, LEN>>>(d_in, d_out);
		singleRotation<<<HLN, HLN>>>(d_in, d_out, paircut);
		hipDeviceSynchronize();
		hipMemcpy(h_in, d_out, B2LEN, hipMemcpyDeviceToHost);
		triggerPaircut();
	}

	auto err = hipGetLastError();
	while (hipSuccess != err) {
		printf("Error %d: %s\n", err, hipGetErrorString(err));
		err = hipGetLastError();
	}
	
	
}

void freeArrays() {
	hipFree(d_in);
	hipFree(d_out);
}

int* get_h_in() {
	return (int*)h_in;
}